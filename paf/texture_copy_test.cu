#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include <hipfft/hipfft.h>

using std::cout;
using std::endl;

#define XSIZE 7
#define YSIZE 128
#define ZSIZE 48

#define cudaCheckError(myerror) {checkGPU((myerror), __FILE__, __LINE__);}

inline void checkGPU(hipError_t code, const char *file, int line) {

    if (code != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(code) << " in file " << file << ", line " << line << std::endl;
        exit(EXIT_FAILURE);
        // TODO: throw exception instead of exiting
    }

}

texture<int2, hipTextureType3D, hipReadModeElementType> tex;
texture<int2, hipTextureType2D, hipReadModeElementType> tex2;

__global__ void arrange(hipfftComplex * __restrict__ out) {

    // xidx is the channel number
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * blockDim.y + threadIdx.y;
    int2 word;

    for (int sample = 0; sample < YSIZE; sample++) {
         word = tex3D(tex, xidx, sample, yidx);
         out[(yidx * XSIZE + xidx) * YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         out[(yidx * XSIZE + xidx) * YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
         //printf("%i, %i, %i, %f, %f, %f, %f\n", xidx, yidx, sample, out[(yidx * XSIZE + xidx) * YSIZE + sample].x, out[(yidx * XSIZE + xidx) * YSIZE + sample].y, out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].x, out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].y);
    }
}

__global__ void arrange2(hipfftComplex * __restrict__ out) {
    // this is currently the ugliest solution I can think of
    // xidx is the channel number
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int2 word;

    for (int sample = 0; sample < YSIZE; sample++) {
         word = tex2D(tex2, xidx, yidx + sample);
         //printf("%i ", sample);
         out[xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         out[xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
         //out[(yidx * XSIZE + xidx) * YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         //out[(yidx * XSIZE + xidx) * YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         //out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         //out[XSIZE * YSIZE * ZSIZE + (yidx * XSIZE + xidx) * YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
         //printf("%i, %i, %i, %f, %f \n", xidx, yidx, sample, out[xidx * 128 + 7 * yidx + sample].x, out[xidx * 128 + 7 * yidx + sample].y);
    }
}


__global__ void arrangebad(char *in, hipfftComplex *out) {

    int xidx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int sample = 0; sample < YSIZE; sample++) {
        out[xidx * YSIZE + sample].x = static_cast<float>(static_cast<short>(in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 7] | (in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 7] << 8)));
        out[xidx * YSIZE + sample].y = static_cast<float>(static_cast<short>(in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 5] | (in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 4] << 8)));
        out[xidx * YSIZE + sample + XSIZE * YSIZE * ZSIZE].x = static_cast<float>(static_cast<short>(in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 3] | (in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 2] << 8)));
        out[xidx * YSIZE + sample + XSIZE * YSIZE * ZSIZE].x = static_cast<float>(static_cast<short>(in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 1] | (in[blockIdx.x * 7 * 128 * 8 + sample * 7 * 8 + threadIdx.x * 8 + 0] << 8)));
    }
}

int main(int argc, char *argv[])
{

    float alloc_elapsed;
    hipEvent_t alloc_start;
    hipEvent_t alloc_end;

    hipEventCreate(&alloc_start);
    hipEventCreate(&alloc_end);

    char *h_in = new char[8 * XSIZE * YSIZE * ZSIZE];

    for (int ii = 0; ii < ZSIZE; ii++) {
        for (int jj = 0; jj < YSIZE; jj++) {
            for (int kk = 0; kk < 8 * XSIZE; kk++) {
                h_in[ii * XSIZE * YSIZE * 8 + jj * XSIZE * 8 + kk] = jj;
            }
        }
    }

    hipfftComplex *d_out;
    cudaCheckError(hipMalloc((void**)&d_out, 2 * XSIZE * YSIZE * ZSIZE * sizeof(hipfftComplex)));

    hipChannelFormatDesc cdesc;
    hipExtent volume;
    hipMemcpy3DParms params = {0};

    cdesc = hipCreateChannelDesc<int2>();
    cudaCheckError(hipPeekAtLastError());
    volume = make_hipExtent(XSIZE, YSIZE, ZSIZE);
    cudaCheckError(hipPeekAtLastError());
    hipArray *d_array;

    hipEventRecord(alloc_start, 0);
    cudaCheckError(hipMalloc3DArray(&d_array, &cdesc, volume));
    cudaCheckError(hipBindTextureToArray(tex, d_array));
    hipEventRecord(alloc_end, 0);
    hipEventSynchronize(alloc_end);
    hipEventElapsedTime(&alloc_elapsed, alloc_start, alloc_end);

    cout << "3D alloc: " << alloc_elapsed << "ms" << endl;

    params.extent = volume;
    params.dstArray = d_array;
    params.kind = hipMemcpyHostToDevice;
    params.srcPtr = make_hipPitchedPtr((void*)h_in, XSIZE * 8, XSIZE * 8, YSIZE);

    tex.filterMode = hipFilterModePoint;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    float copy_elapsed;
    hipEvent_t copy_start;
    hipEvent_t copy_end;

    hipEventCreate(&copy_start);
    hipEventCreate(&copy_end);

    hipEventRecord(copy_start,0);
    cudaCheckError(hipMemcpy3D(&params));
    hipEventRecord(copy_end, 0);
    hipEventSynchronize(copy_end);
    hipEventElapsedTime(&copy_elapsed, copy_start, copy_end);

    cout << "3D memcpy: " << copy_elapsed << "ms" << endl;

    float exec_elapsed;
    hipEvent_t exec_start;
    hipEvent_t exec_end;

    hipEventCreate(&exec_start);
    hipEventCreate(&exec_end);   

    dim3 nblocks(1,1,1); 
    dim3 nthreads(XSIZE, ZSIZE, 1);

    hipEventRecord(exec_start, 0);
    arrange<<<nblocks, nthreads, 0>>>(d_out);    
    cudaCheckError(hipPeekAtLastError());
    hipEventRecord(exec_end, 0);
    hipEventSynchronize(exec_end);
    hipEventElapsedTime(&exec_elapsed, exec_start, exec_end);

    cout << "3D exec: " << exec_elapsed << "ms" << endl;

    hipDeviceSynchronize();
    hipUnbindTexture(tex);

    hipFreeArray(d_array);

    // ###################
    // 2D 'IMPLEMENTATION'
    // ###################

    hipArray *d_array2;
    hipEventRecord(alloc_start, 0);
    cudaCheckError(hipMallocArray(&d_array2, &cdesc, XSIZE, YSIZE * ZSIZE));
    cudaCheckError(hipBindTextureToArray(tex2, d_array2));
    hipEventRecord(alloc_end, 0);
    hipEventSynchronize(alloc_end);
    hipEventElapsedTime(&alloc_elapsed, alloc_start, alloc_end);

    cout << "2D alloc: " << alloc_elapsed << "ms" << endl;

    tex2.filterMode = hipFilterModePoint;
    tex2.addressMode[0] = hipAddressModeClamp;
    tex2.addressMode[1] = hipAddressModeClamp;

    hipEventRecord(copy_start,0);
    cudaCheckError(hipMemcpyToArray(d_array2, 0, 0, h_in, 8 * XSIZE * YSIZE * ZSIZE, hipMemcpyHostToDevice));
    hipEventRecord(copy_end, 0);
    hipEventSynchronize(copy_end);
    hipEventElapsedTime(&copy_elapsed, copy_start, copy_end);

    cout << "2D memcpy: " << copy_elapsed << "ms" << endl;

    dim3 nblocks2(1, ZSIZE, 1);
    dim3 nthreads2(XSIZE,1,1);

    hipEventRecord(exec_start, 0);
    arrange2<<<nblocks2, nthreads2, 0>>>(d_out);
    hipDeviceSynchronize();
    cudaCheckError(hipPeekAtLastError());
    hipEventRecord(exec_end, 0);
    hipEventSynchronize(exec_end);
    hipEventElapsedTime(&exec_elapsed, exec_start, exec_end);

    cout << "2D exec: " << exec_elapsed << "ms" << endl;

    hipfftComplex *h_out = new hipfftComplex[2 * XSIZE * YSIZE * ZSIZE];
    cudaCheckError(hipMemcpy(h_out, d_out, 2 * XSIZE * YSIZE * ZSIZE * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    //for (int ii = 0; ii < YSIZE; ii++)
    //    cout << h_out[ii].x << " + i * " << h_out[ii].y << endl;   

    hipDeviceSynchronize();
    hipUnbindTexture(tex2);

    hipFreeArray(d_array2);

    char *d_in;

    hipEventRecord(alloc_start, 0);    
    cudaCheckError(hipMalloc((void**)&d_in, 8 * XSIZE * YSIZE * ZSIZE));
    hipEventRecord(alloc_end, 0);
    hipEventSynchronize(alloc_end);
    hipEventElapsedTime(&alloc_elapsed, alloc_start, alloc_end);

    cout << "Simple device alloc: " << alloc_elapsed << "ms" << endl;

    hipEventRecord(copy_start,0);
    cudaCheckError(hipMemcpy(d_in, h_in, 8 * XSIZE * YSIZE * ZSIZE, hipMemcpyHostToDevice));
    hipEventRecord(copy_end, 0);
    hipEventSynchronize(copy_end);
    hipEventElapsedTime(&copy_elapsed, copy_start, copy_end);

    cout << "Simple device  memcpy: " << copy_elapsed << "ms" << endl;

    dim3 nblocks3(48, 1, 1);
    dim3 nthreads3(7, 1, 1);

    hipEventRecord(exec_start, 0);
    arrangebad<<<nblocks3, nthreads3>>>(d_in, d_out);
    cudaCheckError(hipPeekAtLastError());
    hipEventRecord(exec_end, 0);
    hipEventSynchronize(exec_end);
    hipEventElapsedTime(&exec_elapsed, exec_start, exec_end);

    cout << "Simple exec: " << exec_elapsed << "ms" << endl;
    

    hipFree(d_in);
    hipFree(d_out); 
    delete [] h_in;

    return 0;
}
